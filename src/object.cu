#include "hip/hip_runtime.h"
//matrix_add.cu
//这个文件不推荐直接拷贝，最好一行一行地抄下来
#include <stdio.h>

#define N 32  //这里定义了矩阵的阶级，这里用一个32x32的方形矩阵做例子
//想要自己Debug的时候，遇到问题，可以先把N的值设置小一些，比如4

/*
 * 本次的核函数，三个参数分别是两个NxN的输入矩阵和一个NxN的输出矩阵
 */
__global__ void calculate_object(const double x[][N],
    const double y[][N],
    const double z[][N],
    const double v[][N],
    const double a[][N],
    const double *interval) {
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    double v_delt =  a[idx][idy] * (*interval);
    double v_new = v[idx][idy] + v_delt;
    double s_new = v[idx][idy] * (*interval) + a[idx][idy]*(*interval)*(*interval) / 2.0;
}

int main(void) {

  double *h_x, *h_y, *h_z, *h_v, *h_a;
  double *dev_x, *dev_y, *dev_z, *dev_v, *dev_a;
  double *h_interval;
  double *dev_interval;

  //这里把一个block的里面的线程排列定义成二维的，这样会有两个维度的索引值，x和y
  dim3 threads_in_block (N, N);
  //err这个值是用来检查cuda的函数是否正常运行的
  hipError_t err = hipSuccess;

  h_x = (double *)malloc(sizeof(double) * N * N);
  h_y = (double *)malloc(sizeof(double) * N * N);
  h_z = (double *)malloc(sizeof(double) * N * N);
  h_v = (double *)malloc(sizeof(double) * N * N);
  h_a = (double *)malloc(sizeof(double) * N * N);

  h_interval = (double*)malloc(sizeof(double));
  
  if (h_x == NULL || h_y == NULL || h_z == NULL
      || h_v == NULL || h_a == NULL) {
    fprintf(stderr, "Malloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_x, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;

  }
  err = hipMalloc((void **)&dev_y, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_z, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_v, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_a, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_interval, sizeof(double));
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }

  for (int i = 0; i < N * N; i++) {
    h_x[i] = 2 * i + 1.0;
    h_y[i] = -1 * i + 5.0;
    h_z[i] = -1 * i + 3.0;
    h_v[i] = -1 * i + 4.0;
    h_a[i] = -1 * i + 8.0;
  }
  h_interval = 3.0;

  err = hipMemcpy(dev_x, h_x, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_y, h_y, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_z, h_z, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(dev_v, h_v, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(dev_a, h_a, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_interval, h_interval, sizeof(double), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  calculate_object<<<1, threads_in_block>>>((double (*)[N])dev_x, (double (*)[N])dev_y, (double (*)[N])dev_z,
      (double (*)[N])dev_v, (double (*)[N])dev_a, (double*)interval);

  /*err = hipMemcpy(h_c, dev_c, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  for (int i = 0; i < N * N; i++) {
    if (h_a[i] + h_b[i] != h_c[i]) {
      fprintf(stderr, "a[%d]%d + b[%d]%d != c[%d]%d.\n", i, h_a[i], i, h_b[i], i, h_c[i]);
      return -1;
    }
  }*/

  printf("done.\n");
  return 0;

}

