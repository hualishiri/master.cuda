#include "hip/hip_runtime.h"
//matrix_add.cu
//这个文件不推荐直接拷贝，最好一行一行地抄下来
#include <stdio.h>
#include <sys/time.h>
#include <time.h>

#define N 32  //这里定义了矩阵的阶级，这里用一个32x32的方形矩阵做例子
//想要自己Debug的时候，遇到问题，可以先把N的值设置小一些，比如4

/*
 * 本次的核函数，三个参数分别是两个NxN的输入矩阵和一个NxN的输出矩阵
 */
__global__ void calculate_object(double x[][N],
    double y[][N],
    double z[][N],
    double v[][N],
    double a[][N],
    double r_x[][N]
    double r_y[][N],
    double r_z[][N],
    double r_v[][N],
    double r_a[][N],
    double *interval) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  for (int i=0; i!=1000000; ++i) {
    double r_a[idx][idy] =  a[idx][idy] * (*interval);
    double r_v[idx][idy] = v[idx][idy] + a[idx][idy];
    double r_x[idx][idy] = v[idx][idy] * (*interval) + a[idx][idy]*(*interval)*(*interval) / 2.0;
  }
}

__host__ void host_calculate_object(double x[][N],
    double y[][N],
    double z[][N],
    double v[][N],
    double a[][N],
    double r_x[][N]
    double r_y[][N],
    double r_z[][N],
    double r_v[][N],
    double r_a[][N],
    double *interval) {
  for (int k=0; k!=1000000; ++k) {
    for (int i=0; i!=N; ++i) {
      for (int j=0; j!=N; ++j) {
        r_a[i][j] = a[i][j] * (*interval);
        double r_v[i][j] = v[i][j] + r_a[i][j];
        double r_x[i][j] = v[i][j] * a[i][j] +
          a[i][j] * (*interval) * (*interval) / 2.0;
      }
    }
  }
}

int main(void) {
  struct timeval start, end, host_start, host_end;
  double elapsed_time, host_elapsed_time;

  double *h_x, *h_y, *h_z, *h_v, *h_a;
  double *h_r_x, *h_r_y, *h_r_z, *h_r_v, *h_r_a;

  double *dev_x, *dev_y, *dev_z, *dev_v, *dev_a;
  double *dev_r_x, *dev_r_y, *dev_r_z, *dev_r_v, *dev_r_a;
  double *h_interval;
  double *dev_interval;

  gettimeofday(&start, NULL);

  //这里把一个block的里面的线程排列定义成二维的，这样会有两个维度的索引值，x和y
  dim3 threads_in_block (N, N);
  //err这个值是用来检查cuda的函数是否正常运行的
  hipError_t err = hipSuccess;

  h_x = (double *)malloc(sizeof(double) * N * N);
  h_y = (double *)malloc(sizeof(double) * N * N);
  h_z = (double *)malloc(sizeof(double) * N * N);
  h_v = (double *)malloc(sizeof(double) * N * N);
  h_a = (double *)malloc(sizeof(double) * N * N);

  h_r_x = (double *)malloc(sizeof(double) * N * N);
  h_r_y = (double *)malloc(sizeof(double) * N * N);
  h_r_z = (double *)malloc(sizeof(double) * N * N);
  h_r_v = (double *)malloc(sizeof(double) * N * N);
  h_r_a = (double *)malloc(sizeof(double) * N * N);

  h_interval = (double*)malloc(sizeof(double));

  if (h_x == NULL || h_y == NULL || h_z == NULL
      || h_v == NULL || h_a == NULL) {
    fprintf(stderr, "Malloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_x, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_y, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_z, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_v, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_a, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_r_x, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_r_y, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_r_z, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_r_v, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }
  err = hipMalloc((void **)&dev_r_a, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_interval, sizeof(double));
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed.\n");
    return -1;
  }

  for (int i = 0; i < N * N; i++) {
    h_x[i] = 2 * i + 1.0;
    h_y[i] = -1 * i + 5.0;
    h_z[i] = -1 * i + 3.0;
    h_v[i] = -1 * i + 4.0;
    h_a[i] = -1 * i + 8.0;
    h_r_x[i] = 2 * i + 1.0;
    h_r_y[i] = -1 * i + 5.0;
    h_r_z[i] = -1 * i + 3.0;
    h_r_v[i] = -1 * i + 4.0;
    h_r_a[i] = -1 * i + 8.0;
  }
  *h_interval = 3.0;

  err = hipMemcpy(dev_x, h_x, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_y, h_y, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_z, h_z, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(dev_v, h_v, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(dev_a, h_a, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_interval, h_interval, sizeof(double), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  calculate_object<<<1, threads_in_block>>>(
      (double (*)[N])dev_x,
      (double (*)[N])dev_y,
      (double (*)[N])dev_z,
      (double (*)[N])dev_v,
      (double (*)[N])dev_a,
      (double (*)[N])dev_r_x,
      (double (*)[N])dev_r_y,
      (double (*)[N])dev_r_z,
      (double (*)[N])dev_r_v,
      (double (*)[N])dev_r_a,
      (double*)dev_interval);


  err = hipMemcpy(h_r_x,dev_r_x, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(h_r_y,dev_r_y, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(h_r_z,dev_r_z, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(h_r_v,dev_r_v, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(h_r_a,dev_r_a, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(h_r_interval,dev_r_interval, sizeof(double), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
  }


  gettimeofday(&end, NULL);
  elapsed_time = (end.tv_sec - start.tv_sec) * 1000.0;
  elapsed_time += (end.tv_usec - start.tv_usec) / 1000.0;

  /*err = hipMemcpy(h_c, dev_c, sizeof(double) * N * N, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed.\n");
    return -1;
    }

    for (int i = 0; i < N * N; i++) {
    if (h_a[i] + h_b[i] != h_c[i]) {
    fprintf(stderr, "a[%d]%d + b[%d]%d != c[%d]%d.\n", i, h_a[i], i, h_b[i], i, h_c[i]);
    return -1;
    }
    }*/

  gettimeofday(&host_start, NULL);
  host_calculate_object(
      (double (*)[N])h_x,
      (double (*)[N])h_y,
      (double (*)[N])h_z,
      (double (*)[N])h_v,
      (double (*)[N])h_a,
      (double (*)[N])h_r_x,
      (double (*)[N])h_r_y,
      (double (*)[N])h_r_z,
      (double (*)[N])h_r_v,
      (double (*)[N])h_r_a,
      h_interval);
  gettimeofday(&host_end, NULL);
  host_elapsed_time = (host_end.tv_sec - host_start.tv_sec) * 1000.0;
  host_elapsed_time += (host_end.tv_usec - host_start.tv_usec) / 1000.0;

  printf("cuda finished in %f milliseconds.\n", elapsed_time);
  printf("host finished in %f milliseconds.\n", host_elapsed_time);
  printf("speed up rate: %f\n", host_elapsed_time / elapsed_time);

  printf("done.\n");
  return 0;
}

