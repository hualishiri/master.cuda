//matrix_add.cu
//这个文件不推荐直接拷贝，最好一行一行地抄下来

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <time.h>

#define N 32  //这里定义了矩阵的阶级，这里用一个32x32的方形矩阵做例子
//想要自己Debug的时候，遇到问题，可以先把N的值设置小一些，比如4

/*
 * 本次的核函数，三个参数分别是两个NxN的输入矩阵和一个NxN的输出矩阵
 */
__global__ void calculate_object(const double x[][N],
    const double y[][N],
    const double z[][N],
    const double v[][N],
    const double a[][N],
    const double *interval) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  for (int i=0; i!=1000000; ++i) {
    double v_delt =  a[idx][idy] * (*interval);
    double v_new = v[idx][idy] + v_delt;
    double s_new = v[idx][idy] * (*interval) + a[idx][idy]*(*interval)*(*interval) / 2.0;
  }
}

__host__ void host_calculate_object(const double x[][N],
    const double y[][N],
    const double z[][N],
    const double v[][N],
    const double a[][N],
    const double *interval) {
  for (int k=0; k!=1000000; ++k) {
    for (int i=0; i!=N; ++i) {
      for (int j=0; j!=N; ++j) {
        double v_delt = a[i][j] * (*interval);
        double v_new = v[i][j] + v_delt;
        double s_new = v[i][j] * (*interval) +
          a[i][j] * (*interval) * (*interval) / 2.0;
      }
    }
  }
}

int main(void) {
  struct timeval start, end, host_start, host_end;
  double elapsed_time, host_elapsed_time;

  double *h_x, *h_y, *h_z, *h_v, *h_a;
  double *dev_x, *dev_y, *dev_z, *dev_v, *dev_a;
  double *h_interval;
  double *dev_interval;

  gettimeofday(&start, NULL);

  //这里把一个block的里面的线程排列定义成二维的，这样会有两个维度的索引值，x和y
  dim3 threads_in_block (N, N);
  //err这个值是用来检查cuda的函数是否正常运行的
  hipError_t err = hipSuccess;

  h_x = (double *)malloc(sizeof(double) * N * N);
  h_y = (double *)malloc(sizeof(double) * N * N);
  h_z = (double *)malloc(sizeof(double) * N * N);
  h_v = (double *)malloc(sizeof(double) * N * N);
  h_a = (double *)malloc(sizeof(double) * N * N);

  h_interval = (double*)malloc(sizeof(double));

  if (h_x == NULL || h_y == NULL || h_z == NULL
      || h_v == NULL || h_a == NULL) {
    fprintf(stderr, "Malloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_x, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;

  }
  err = hipMalloc((void **)&dev_y, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_z, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_v, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_a, sizeof(double) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;
  }

  err = hipMalloc((void **)&dev_interval, sizeof(double));
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;
  }

  for (int i = 0; i < N * N; i++) {
    h_x[i] = 2 * i + 1.0;
    h_y[i] = -1 * i + 5.0;
    h_z[i] = -1 * i + 3.0;
    h_v[i] = -1 * i + 4.0;
    h_a[i] = -1 * i + 8.0;
  }
  *h_interval = 3.0;

  err = hipMemcpy(dev_x, h_x, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_y, h_y, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_z, h_z, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(dev_v, h_v, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;
  }
  err = hipMemcpy(dev_a, h_a, sizeof(double) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;
  }

  err = hipMemcpy(dev_interval, h_interval, sizeof(double), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;
  }

  calculate_object<<<1, threads_in_block>>>((double (*)[N])dev_x, (double (*)[N])dev_y, (double (*)[N])dev_z,
      (double (*)[N])dev_v, (double (*)[N])dev_a, (double*)dev_interval);

  gettimeofday(&end, NULL);
  elapsed_time = (end.tv_sec - start.tv_sec) * 1000.0;
  elapsed_time += (end.tv_usec - start.tv_usec) / 1000.0;

  /*err = cudaMemcpy(h_c, dev_c, sizeof(double) * N * N, cudaMemcpyDeviceToHost);
    if (err != cudaSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;
    }

    for (int i = 0; i < N * N; i++) {
    if (h_a[i] + h_b[i] != h_c[i]) {
    fprintf(stderr, "a[%d]%d + b[%d]%d != c[%d]%d.\n", i, h_a[i], i, h_b[i], i, h_c[i]);
    return -1;
    }
    }*/

  gettimeofday(&host_start, NULL);
  host_calculate_object((double (*)[N])h_x, (double (*)[N])h_y, (double (*)[N])h_z,
      (double (*)[N])h_v, (double (*)[N])h_z, h_interval);
  gettimeofday(&host_end, NULL);
  host_elapsed_time = (host_end.tv_sec - host_start.tv_sec) * 1000.0;
  host_elapsed_time += (host_end.tv_usec - host_start.tv_usec) / 1000.0;

  printf("cuda finished in %f milliseconds.\n", elapsed_time);
  printf("host finished in %f milliseconds.\n", host_elapsed_time);
  printf("speed up rate: %f\n", host_elapsed_time / elapsed_time);

  printf("done.\n");
  return 0;

}

