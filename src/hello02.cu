//matrix_add.cu
//这个文件不推荐直接拷贝，最好一行一行地抄下来

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32  //这里定义了矩阵的阶级，这里用一个32x32的方形矩阵做例子
//想要自己Debug的时候，遇到问题，可以先把N的值设置小一些，比如4

/*
 * 本次的核函数，三个参数分别是两个NxN的输入矩阵和一个NxN的输出矩阵
 */
__global__ void matrix_add(const int a[][N], const int b[][N], int c[][N]) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  c[idx][idy] = a[idx][idy] + b[idx][idy];

}

int main(void) {

  int *h_a, *h_b, *h_c;
  int *dev_a, *dev_b, *dev_c;

  //这里把一个block的里面的线程排列定义成二维的，这样会有两个维度的索引值，x和y
  dim3 threads_in_block (N, N);
  //err这个值是用来检查cuda的函数是否正常运行的
  hipError_t err = hipSuccess;

  h_a = (int *)malloc(sizeof(int) * N * N);
  h_b = (int *)malloc(sizeof(int) * N * N);
  h_c = (int *)malloc(sizeof(int) * N * N);

  if (h_a == NULL || h_b == NULL || h_c == NULL) {
    fprintf(stderr, "Malloc() failed.\n");
    return -1;

  }

  err = hipMalloc((void **)&dev_a, sizeof(int) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;

  }
  err = hipMalloc((void **)&dev_b, sizeof(int) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;

  }
  err = hipMalloc((void **)&dev_c, sizeof(int) * N * N);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() failed.\n");
    return -1;

  }

  for (int i = 0; i < N * N; i++) {
    h_a[i] = 2 * i + 1;
    h_b[i] = -1 * i + 5;

  }

  err = hipMemcpy(dev_a, h_a, sizeof(int) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;

  }
  err = hipMemcpy(dev_b, h_b, sizeof(int) * N * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;

  }

  matrix_add<<<1, threads_in_block>>>((int (*)[N])dev_a, (int (*)[N])dev_b, (int (*)[N])dev_c);

  err = hipMemcpy(h_c, dev_c, sizeof(int) * N * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "cudaMemcpy() failed.\n");
    return -1;

  }

  for (int i = 0; i < N * N; i++) {
    if (h_a[i] + h_b[i] != h_c[i]) {
      fprintf(stderr, "a[%d]%d + b[%d]%d != c[%d]%d.\n", i, h_a[i], i, h_b[i], i, h_c[i]);
      return -1;

    }

  }

  printf("done.\n");
  return 0;

}

